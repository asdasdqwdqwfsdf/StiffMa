
#include <hip/hip_runtime.h>
/*=========================================================================
 *
 ** IndexVectorGPU - Row/column indices of the lower triangular sparse matrix K (VECTOR)
 *
 *
 ** DATA INPUT
 * 			elements[8][nel]      // Conectivity matrix of the mesh [gpuArray(uint32(elements))]
 *
 ** DATA OUTPUT
 *			iK[300*nel]           // Row indices of the lower-triangular part of ke
 *			jK[300*nel]           // Colummn indices of lower-triangular part of ke
 *
 ** COMPILATION LINUX (Terminal)
 *          sudo nano ~/.bashrc
 *          export PATH=/usr/local/cuda-9.2/bin${PATH:+:${PATH}}
 * 			nvcc -ptx IndexVectorGPU.cu 
 *
 ** COMPILATION WINDOWS (Terminal)
 * 			nvcc -ptx IndexVectorGPU.cu 
 *
 ** MATLAB KERNEL CREATION
 *			kernel = parallel.gpu.CUDAKernel('IndexVectorGPU.ptx', 'IndexVectorGPU.cu');
 *
 ** MATLAB KERNEL CONFIGURATION
 *          kernel.ThreadBlockSize = [512, 1, 1];
 *          kernel.GridSize = [ceil(nel/kernel.ThreadBlockSize(1)), 1, 1];
 *
 ** MATLAB CALL
 *			Out = feval(kernel, DATA INPUT + DATA OUTPUT);
 *          [iK, jK] = feval(kernel, elements, nel, gpuArray.zeros(300*nel,1,'uint32'), gpuArray.zeros(300*nel,1,'uint32'));
 *
 ** TRANSFER DATA FROM CPU TO GPU MEMORY (if necessary)
 *			Out_cpu = gather(Out);
 *
 ** This function was developed by:
 *          Francisco Javier Ramirez-Gil
 *          Universidad Nacional de Colombia - Medellin
 *          Department of Mechanical Engineering
 *
 ** Please cite this code as:
 *
 ** Date & version
 *      22/11/2018.
 *      V 1.0
 *
 * ======================================================================*/

__global__ void IndexScalarGPU(const unsigned int *elements,
                               const unsigned int nel,
                               unsigned int *iK, unsigned int *jK ){
    // CUDA kernel to compute row/column indices of tril(K) (VECOTR)
    
    int tid = blockDim.x * blockIdx.x + threadIdx.x;    // Thread ID
    unsigned int i, j, temp, idx, n[8], dof[24];        // General indices
    
    if (tid < nel)	{                                   // Parallel computation
        
        // Extract the nodes associated with element 'e' (=tid)
        for (i=0; i<8; i++) {n[i] = elements[i+8*tid];}
        
        // Extract the global dof associated with element 'e' (=tid)
        for (i=0; i<8; i++) {
            dof[3*i  ] = 3*n[i] - 2;
            dof[3*i+1] = 3*n[i] - 1;
            dof[3*i+2] = 3*n[i];
        }
        
        temp = 0;
        for (j=0; j<24; j++){
            for (i=j; i<24; i++){
                idx = temp + i + 300*tid;
                if (dof[i] > dof[j]){
                    iK[idx] = dof[i];
                    jK[idx] = dof[j]; }
                else {
                    iK[idx] = dof[j];
                    jK[idx] = dof[i];
                }
            }
            temp += i-j-1;
        }
    }
}
