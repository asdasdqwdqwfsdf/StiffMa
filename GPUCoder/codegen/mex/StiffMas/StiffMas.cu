#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * StiffMas.cu
 *
 * Code generation for function 'StiffMas'
 *
 */

/* Include files */
#include "StiffMas.h"
#include "MWCudaDimUtility.h"
#include "StiffMas_emxutil.h"
#include "rt_nonfinite.h"
#include <math.h>
#include <string.h>

/* Function Declarations */
static __global__ void StiffMas_kernel1(const emxArray_real_T *nodes, const
  int32_T e, const emxArray_uint32_T *elements, real_T X[24]);
static __global__ void StiffMas_kernel10(const int32_T jp1j, const real_T Jac[9],
  const int32_T jy, const real_T L[24], const int32_T jA, real_T B[24]);
static __global__ void StiffMas_kernel11(const real_T B[24], real_T b_B[64]);
static __global__ void StiffMas_kernel12(const real_T B[64], const real_T smax,
  const emxArray_real_T *Ke, const int32_T e, real_T b_Ke[64]);
static __global__ void StiffMas_kernel13(const real_T Ke[64], const int32_T e,
  emxArray_real_T *b_Ke);
static __global__ void StiffMas_kernel2(const emxArray_uint32_T *elements, const
  int32_T e, uint32_T ind[64]);
static __global__ void StiffMas_kernel3(const uint32_T ind[64], const int32_T e,
  emxArray_real_T *Ke, emxArray_uint32_T *jK, emxArray_uint32_T *iK);
static __global__ void StiffMas_kernel4(const real_T X[24], const real_T L[24],
  real_T Jac[9]);
static __global__ void StiffMas_kernel5(const real_T Jac[9], real_T x[9]);
static __global__ void StiffMas_kernel6(int8_T ipiv[3]);
static __global__ void StiffMas_kernel7(const real_T x[9], real_T *detJ);
static __global__ void StiffMas_kernel8(const int32_T jy, const int32_T jp1j,
  real_T Jac[9]);
static __global__ void StiffMas_kernel9(const int32_T jy, const int32_T jp1j,
  real_T Jac[9]);
static __inline__ __device__ real_T atomicOpreal_T(real_T *address, real_T value);
static void gpuEmxFree_real_T(emxArray_real_T *inter);
static void gpuEmxFree_uint32_T(emxArray_uint32_T *inter);
static void gpuEmxMemcpyCpuToGpu_real_T(const emxArray_real_T *cpu,
  emxArray_real_T *inter, emxArray_real_T *gpu);
static void gpuEmxMemcpyCpuToGpu_uint32_T(const emxArray_uint32_T *cpu,
  emxArray_uint32_T *inter, emxArray_uint32_T *gpu);
static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T
  *inter);
static void gpuEmxMemcpyGpuToCpu_uint32_T(emxArray_uint32_T *cpu,
  emxArray_uint32_T *inter);
static void gpuEmxReset_real_T(emxArray_real_T *inter);
static void gpuEmxReset_uint32_T(emxArray_uint32_T *inter);
static __inline__ __device__ real_T shflDown2(real_T in1, uint32_T offset,
  uint32_T mask);
static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
  uint32_T mask);
static __device__ real_T workGroupReduction(real_T val, uint32_T mask, uint32_T
  numActiveWarps);

/* Function Definitions */
static __global__ __launch_bounds__(32, 1) void StiffMas_kernel1(const
  emxArray_real_T *nodes, const int32_T e, const emxArray_uint32_T *elements,
  real_T X[24])
{
  uint32_T threadId;
  int32_T ibmat;
  int32_T jcol;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  ibmat = static_cast<int32_T>((threadId % 8U));
  jcol = static_cast<int32_T>(((threadId - static_cast<uint32_T>(ibmat)) / 8U));
  if (jcol < 3) {
    X[ibmat + (jcol << 3)] = nodes->data[(static_cast<int32_T>(elements->data[e
      + elements->size[0] * ibmat]) + nodes->size[0] * jcol) - 1];
  }
}

static __global__ __launch_bounds__(32, 1) void StiffMas_kernel10(const int32_T
  jp1j, const real_T Jac[9], const int32_T jy, const real_T L[24], const int32_T
  jA, real_T B[24])
{
  uint32_T threadId;
  real_T d;
  int32_T jcol;
  real_T d1;
  real_T d2;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  jcol = static_cast<int32_T>(threadId);
  if (jcol < 8) {
    d = L[(jA + 3 * jcol) - 1];
    d1 = L[(jy + 3 * jcol) - 1] - d * Jac[jy - 1];
    d2 = ((L[(jp1j + 3 * jcol) - 1] - d * Jac[jp1j - 1]) - d1 * Jac[jp1j + 2]) /
      Jac[jp1j + 5];
    B[3 * jcol + 2] = d2;
    d -= d2 * Jac[jA + 5];
    d1 -= d2 * Jac[jy + 5];
    d1 /= Jac[jy + 2];
    B[3 * jcol + 1] = d1;
    d -= d1 * Jac[jA + 2];
    d /= Jac[jA - 1];
    B[3 * jcol] = d;
  }
}

static __global__ __launch_bounds__(64, 1) void StiffMas_kernel11(const real_T
  B[24], real_T b_B[64])
{
  uint32_T threadId;
  real_T d;
  int32_T ibmat;
  int32_T jcol;
  int32_T itilerow;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  itilerow = static_cast<int32_T>((threadId % 8U));
  jcol = static_cast<int32_T>(((threadId - static_cast<uint32_T>(itilerow)) / 8U));
  if (jcol < 8) {
    d = 0.0;
    for (ibmat = 0; ibmat < 3; ibmat++) {
      d += B[ibmat + 3 * jcol] * B[ibmat + 3 * itilerow];
    }

    b_B[jcol + (itilerow << 3)] = d;
  }
}

static __global__ __launch_bounds__(64, 1) void StiffMas_kernel12(const real_T
  B[64], const real_T smax, const emxArray_real_T *Ke, const int32_T e, real_T
  b_Ke[64])
{
  uint32_T threadId;
  int32_T ibmat;
  int32_T jcol;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  ibmat = static_cast<int32_T>((threadId % 8U));
  jcol = static_cast<int32_T>(((threadId - static_cast<uint32_T>(ibmat)) / 8U));
  if (jcol < 8) {
    b_Ke[ibmat + (jcol << 3)] = Ke->data[(ibmat + (jcol << 3)) + (e << 6)] +
      smax * B[ibmat + (jcol << 3)];
  }
}

static __global__ __launch_bounds__(64, 1) void StiffMas_kernel13(const real_T
  Ke[64], const int32_T e, emxArray_real_T *b_Ke)
{
  uint32_T threadId;
  int32_T ibmat;
  int32_T jcol;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  ibmat = static_cast<int32_T>((threadId % 8U));
  jcol = static_cast<int32_T>(((threadId - static_cast<uint32_T>(ibmat)) / 8U));
  if (jcol < 8) {
    b_Ke->data[(ibmat + (jcol << 3)) + (e << 6)] = Ke[ibmat + (jcol << 3)];
  }
}

static __global__ __launch_bounds__(64, 1) void StiffMas_kernel2(const
  emxArray_uint32_T *elements, const int32_T e, uint32_T ind[64])
{
  uint32_T threadId;
  int32_T jcol;
  int32_T itilerow;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  itilerow = static_cast<int32_T>((threadId % 8U));
  jcol = static_cast<int32_T>(((threadId - static_cast<uint32_T>(itilerow)) / 8U));
  if (jcol < 8) {
    ind[(jcol << 3) + itilerow] = elements->data[e + elements->size[0] * jcol];
  }
}

static __global__ __launch_bounds__(64, 1) void StiffMas_kernel3(const uint32_T
  ind[64], const int32_T e, emxArray_real_T *Ke, emxArray_uint32_T *jK,
  emxArray_uint32_T *iK)
{
  uint32_T threadId;
  int32_T ibmat;
  int32_T jcol;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  ibmat = static_cast<int32_T>((threadId % 8U));
  jcol = static_cast<int32_T>(((threadId - static_cast<uint32_T>(ibmat)) / 8U));
  if (jcol < 8) {
    iK->data[(ibmat + (jcol << 3)) + (e << 6)] = ind[jcol + (ibmat << 3)];
    jK->data[(ibmat + (jcol << 3)) + (e << 6)] = ind[ibmat + (jcol << 3)];
    Ke->data[(ibmat + (jcol << 3)) + (e << 6)] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void StiffMas_kernel4(const real_T X
  [24], const real_T L[24], real_T Jac[9])
{
  uint32_T threadId;
  real_T d;
  int32_T ibmat;
  int32_T jcol;
  int32_T itilerow;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  itilerow = static_cast<int32_T>((threadId % 3U));
  jcol = static_cast<int32_T>(((threadId - static_cast<uint32_T>(itilerow)) / 3U));
  if (jcol < 3) {
    d = 0.0;
    for (ibmat = 0; ibmat < 8; ibmat++) {
      d += L[jcol + 3 * ibmat] * X[ibmat + (itilerow << 3)];
    }

    Jac[jcol + 3 * itilerow] = d;
  }
}

static __global__ __launch_bounds__(32, 1) void StiffMas_kernel5(const real_T
  Jac[9], real_T x[9])
{
  uint32_T threadId;
  int32_T jcol;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  jcol = static_cast<int32_T>(threadId);
  if (jcol < 9) {
    /*  Jacobian matrix */
    /* 'Hex8scalars:40' detJ = det(Jac); */
    x[jcol] = Jac[jcol];
  }
}

static __global__ __launch_bounds__(32, 1) void StiffMas_kernel6(int8_T ipiv[3])
{
  uint32_T threadId;
  int32_T jcol;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  jcol = static_cast<int32_T>(threadId);
  if (jcol < 3) {
    ipiv[jcol] = static_cast<int8_T>((jcol + 1));
  }
}

static __global__ __launch_bounds__(32, 1) void StiffMas_kernel7(const real_T x
  [9], real_T *detJ)
{
  uint32_T idx;
  real_T tmpRed0;
  uint32_T threadStride;
  uint32_T threadId;
  uint32_T thBlkId;
  uint32_T mask;
  uint32_T numActiveThreads;
  uint32_T numActiveWarps;
  uint32_T blockStride;
  int32_T m;
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<uint32_T>(mwGetThreadsPerBlock());
  tmpRed0 = 1.0;
  numActiveThreads = blockStride;
  if (mwIsLastBlock()) {
    m = static_cast<int32_T>((3U % blockStride));
    if (static_cast<uint32_T>(m) > 0U) {
      numActiveThreads = static_cast<uint32_T>(m);
    }
  }

  numActiveWarps = ((numActiveThreads + warpSize) - 1U) / warpSize;
  if (threadId <= 2U) {
    tmpRed0 = x[static_cast<int32_T>(threadId) + 3 * static_cast<int32_T>
      (threadId)];
  }

  mask = __ballot_sync(MAX_uint32_T, threadId <= 2U);
  for (idx = threadId + threadStride; idx <= 2U; idx += threadStride) {
    tmpRed0 *= x[static_cast<int32_T>(idx) + 3 * static_cast<int32_T>(idx)];
  }

  tmpRed0 = workGroupReduction(tmpRed0, mask, numActiveWarps);
  if (thBlkId == 0U) {
    atomicOpreal_T(&detJ[0], tmpRed0);
  }
}

static __global__ __launch_bounds__(32, 1) void StiffMas_kernel8(const int32_T
  jy, const int32_T jp1j, real_T Jac[9])
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    Jac[jp1j + 2] /= Jac[jy + 2];
  }
}

static __global__ __launch_bounds__(32, 1) void StiffMas_kernel9(const int32_T
  jy, const int32_T jp1j, real_T Jac[9])
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    Jac[jp1j + 5] -= Jac[jp1j + 2] * Jac[jy + 5];
  }
}

static __inline__ __device__ real_T atomicOpreal_T(real_T *address, real_T value)
{
  unsigned long long int *address_as_up;
  unsigned long long int old;
  unsigned long long int assumed;
  address_as_up = (unsigned long long int *)address;
  old = *address_as_up;
  do {
    assumed = old;
    old = atomicCAS(address_as_up, old, __double_as_longlong(value *
      __longlong_as_double(old)));
  } while (assumed != old);

  return __longlong_as_double(old);
}

static void gpuEmxFree_real_T(emxArray_real_T *inter)
{
  hipFree(inter->data);
  hipFree(inter->size);
}

static void gpuEmxFree_uint32_T(emxArray_uint32_T *inter)
{
  hipFree(inter->data);
  hipFree(inter->size);
}

static void gpuEmxMemcpyCpuToGpu_real_T(const emxArray_real_T *cpu,
  emxArray_real_T *inter, emxArray_real_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  int32_T allocatingSize;
  if (inter->numDimensions < cpu->numDimensions) {
    inter->numDimensions = cpu->numDimensions;
    hipFree(inter->size);
    hipMalloc(&inter->size, inter->numDimensions * sizeof(int32_T));
  } else {
    inter->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  if (inter->allocatedSize < actualSize) {
    if (inter->canFreeData) {
      hipFree(inter->data);
    }

    allocatingSize = cpu->allocatedSize;
    if (allocatingSize < actualSize) {
      allocatingSize = actualSize;
    }

    inter->allocatedSize = allocatingSize;
    inter->canFreeData = true;
    hipMalloc(&inter->data, inter->allocatedSize * sizeof(real_T));
  }

  hipMemcpy(inter->data, cpu->data, actualSize * sizeof(real_T),
             hipMemcpyHostToDevice);
  hipMemcpy(inter->size, cpu->size, cpu->numDimensions * sizeof(int32_T),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu, inter, 32UL, hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyCpuToGpu_uint32_T(const emxArray_uint32_T *cpu,
  emxArray_uint32_T *inter, emxArray_uint32_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  int32_T allocatingSize;
  if (inter->numDimensions < cpu->numDimensions) {
    inter->numDimensions = cpu->numDimensions;
    hipFree(inter->size);
    hipMalloc(&inter->size, inter->numDimensions * sizeof(int32_T));
  } else {
    inter->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  if (inter->allocatedSize < actualSize) {
    if (inter->canFreeData) {
      hipFree(inter->data);
    }

    allocatingSize = cpu->allocatedSize;
    if (allocatingSize < actualSize) {
      allocatingSize = actualSize;
    }

    inter->allocatedSize = allocatingSize;
    inter->canFreeData = true;
    hipMalloc(&inter->data, inter->allocatedSize * sizeof(uint32_T));
  }

  hipMemcpy(inter->data, cpu->data, actualSize * sizeof(uint32_T),
             hipMemcpyHostToDevice);
  hipMemcpy(inter->size, cpu->size, cpu->numDimensions * sizeof(int32_T),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu, inter, 32UL, hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T
  *inter)
{
  int32_T actualSize;
  int32_T i;
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  hipMemcpy(cpu->data, inter->data, actualSize * sizeof(real_T),
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu->size, inter->size, inter->numDimensions * sizeof(int32_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxMemcpyGpuToCpu_uint32_T(emxArray_uint32_T *cpu,
  emxArray_uint32_T *inter)
{
  int32_T actualSize;
  int32_T i;
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  hipMemcpy(cpu->data, inter->data, actualSize * sizeof(uint32_T),
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu->size, inter->size, inter->numDimensions * sizeof(int32_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxReset_real_T(emxArray_real_T *inter)
{
  memset(inter, 0, sizeof(emxArray_real_T));
}

static void gpuEmxReset_uint32_T(emxArray_uint32_T *inter)
{
  memset(inter, 0, sizeof(emxArray_uint32_T));
}

static __inline__ __device__ real_T shflDown2(real_T in1, uint32_T offset,
  uint32_T mask)
{
  int2 tmp;
  tmp = *(int2 *)&in1;
  tmp.x = __shfl_down_sync(mask, tmp.x, offset);
  tmp.y = __shfl_down_sync(mask, tmp.y, offset);
  return *(real_T *)&tmp;
}

static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
  uint32_T mask)
{
  real_T other;
  uint32_T offset;
  uint32_T activeSize;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    other = shflDown2(val, offset, mask);
    if (lane + offset < activeSize) {
      val *= other;
    }

    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }

  return val;
}

static __device__ real_T workGroupReduction(real_T val, uint32_T mask, uint32_T
  numActiveWarps)
{
  __shared__ real_T shared[32];
  uint32_T lane;
  uint32_T widx;
  uint32_T thBlkId;
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  lane = thBlkId % warpSize;
  widx = thBlkId / warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[widx] = val;
  }

  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (widx == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }

  return val;
}

/*
 * function [iK, jK, Ke] = StiffMas(elements,nodes,c)
 */
void StiffMas(const emxArray_uint32_T *elements, const emxArray_real_T *nodes,
              real_T c, emxArray_uint32_T *iK, emxArray_uint32_T *jK,
              emxArray_real_T *Ke)
{
  int32_T i;
  int32_T i1;
  int32_T e;
  real_T L[24];
  real_T Jac[9];
  real_T x[9];
  int8_T ipiv[3];
  int32_T b_i;
  static const real_T dv[8] = { -0.57735026918962584, -0.57735026918962584,
    0.57735026918962584, 0.57735026918962584, -0.57735026918962584,
    -0.57735026918962584, 0.57735026918962584, 0.57735026918962584 };

  static const real_T dv1[8] = { -0.57735026918962584, -0.57735026918962584,
    -0.57735026918962584, -0.57735026918962584, 0.57735026918962584,
    0.57735026918962584, 0.57735026918962584, 0.57735026918962584 };

  static const real_T dv2[8] = { -0.57735026918962584, 0.57735026918962584,
    0.57735026918962584, -0.57735026918962584, -0.57735026918962584,
    0.57735026918962584, 0.57735026918962584, -0.57735026918962584 };

  int32_T j;
  real_T detJ;
  int32_T b_c;
  int32_T jp1j;
  int32_T c_c;
  boolean_T isodd;
  int32_T jA;
  int32_T jy;
  int32_T ix;
  real_T smax;
  real_T s;
  int32_T iy;
  int32_T ijA;
  int32_T b_ijA;
  int32_T initAuxVar;
  emxArray_real_T *gpu_nodes;
  emxArray_uint32_T *gpu_elements;
  real_T (*gpu_X)[24];
  uint32_T (*gpu_ind)[64];
  emxArray_real_T *gpu_Ke;
  emxArray_uint32_T *gpu_jK;
  emxArray_uint32_T *gpu_iK;
  real_T (*gpu_L)[24];
  real_T (*gpu_Jac)[9];
  real_T (*gpu_x)[9];
  int8_T (*gpu_ipiv)[3];
  real_T *gpu_detJ;
  real_T (*gpu_B)[24];
  real_T (*b_gpu_B)[64];
  real_T (*b_gpu_Ke)[64];
  boolean_T Ke_dirtyOnGpu;
  boolean_T jK_dirtyOnGpu;
  boolean_T iK_dirtyOnGpu;
  boolean_T x_dirtyOnGpu;
  boolean_T ipiv_dirtyOnGpu;
  boolean_T detJ_dirtyOnGpu;
  boolean_T nodes_dirtyOnCpu;
  boolean_T elements_dirtyOnCpu;
  boolean_T Ke_dirtyOnCpu;
  boolean_T jK_dirtyOnCpu;
  boolean_T iK_dirtyOnCpu;
  boolean_T x_dirtyOnCpu;
  boolean_T ipiv_dirtyOnCpu;
  emxArray_uint32_T inter_elements;
  emxArray_uint32_T inter_iK;
  emxArray_uint32_T inter_jK;
  emxArray_real_T inter_Ke;
  emxArray_real_T inter_nodes;
  hipMalloc(&b_gpu_Ke, 512UL);
  hipMalloc(&b_gpu_B, 512UL);
  hipMalloc(&gpu_B, 192UL);
  hipMalloc(&gpu_detJ, 8UL);
  hipMalloc(&gpu_ipiv, 3UL);
  hipMalloc(&gpu_x, 72UL);
  hipMalloc(&gpu_Jac, 72UL);
  hipMalloc(&gpu_L, 192UL);
  hipMalloc(&gpu_ind, 256UL);
  hipMalloc(&gpu_X, 192UL);
  hipMalloc(&gpu_nodes, 32UL);
  gpuEmxReset_real_T(&inter_nodes);
  hipMalloc(&gpu_Ke, 32UL);
  gpuEmxReset_real_T(&inter_Ke);
  hipMalloc(&gpu_jK, 32UL);
  gpuEmxReset_uint32_T(&inter_jK);
  hipMalloc(&gpu_iK, 32UL);
  gpuEmxReset_uint32_T(&inter_iK);
  hipMalloc(&gpu_elements, 32UL);
  gpuEmxReset_uint32_T(&inter_elements);
  ipiv_dirtyOnCpu = false;
  x_dirtyOnCpu = false;
  Ke_dirtyOnGpu = false;
  jK_dirtyOnGpu = false;
  iK_dirtyOnGpu = false;
  nodes_dirtyOnCpu = true;
  elements_dirtyOnCpu = true;

  /*  STIFFMAS Create the global stiffness matrix K for a SCALAR problem in SERIAL computing. */
  /*    STIFFMAS(elements,nodes,c) returns a sparse matrix K from finite element */
  /*    analysis of scalar problems in a three-dimensional domain, where "elements" */
  /*    is the connectivity matrix of size nelx8, "nodes" the nodal coordinates of */
  /*    size Nx3, and "c" the material property for a linear isotropic material (scalar). */
  /*  */
  /*    See also STIFFMASS, STIFFMAPS, SPARSE */
  /*  */
  /*    For more information, see the <a href="matlab: */
  /*    web('https://github.com/fjramireg/StiffMa')">StiffMa</a> web site. */
  /*    Written by Francisco Javier Ramirez-Gil, fjramireg@gmail.com */
  /*    Universidad Nacional de Colombia - Medellin */
  /*  	Modified: 05/12/2019. Version: 1.4. Name changed, Doc improved */
  /*  	Modified: 21/01/2019. Version: 1.3 */
  /*    Created:  30/11/2018. Version: 1.0 */
  /*  Initialization */
  /* 'StiffMas:20' dTypeInd = class(elements); */
  /*  Data type (precision) for index computation */
  /* 'StiffMas:21' dTypeKe = class(nodes); */
  /*  Data type (precision) for ke computation */
  /* 'StiffMas:22' nel = size(elements,1); */
  /*  Total number of elements */
  /* 'StiffMas:23' iK = zeros(8,8,nel,dTypeInd); */
  /*  Stores the rows' indices */
  /* 'StiffMas:24' jK = zeros(8,8,nel,dTypeInd); */
  /*  Stores the columns' indices */
  /* 'StiffMas:25' Ke = zeros(8,8,nel,dTypeKe); */
  /*  Stores the NNZ values */
  /*  Add kernelfun pragma to trigger kernel creation */
  /* 'StiffMas:28' coder.gpu.kernelfun; */
  /* 'StiffMas:30' for e = 1:nel */
  i = elements->size[0];
  i1 = iK->size[0] * iK->size[1] * iK->size[2];
  iK->size[0] = 8;
  iK->size[1] = 8;
  iK->size[2] = elements->size[0];
  emxEnsureCapacity_uint32_T(iK, i1);
  iK_dirtyOnCpu = true;
  i1 = jK->size[0] * jK->size[1] * jK->size[2];
  jK->size[0] = 8;
  jK->size[1] = 8;
  jK->size[2] = elements->size[0];
  emxEnsureCapacity_uint32_T(jK, i1);
  jK_dirtyOnCpu = true;
  i1 = Ke->size[0] * Ke->size[1] * Ke->size[2];
  Ke->size[0] = 8;
  Ke->size[1] = 8;
  Ke->size[2] = elements->size[0];
  emxEnsureCapacity_real_T(Ke, i1);
  Ke_dirtyOnCpu = true;
  for (e = 0; e < i; e++) {
    /*  Loop over elements */
    /* 'StiffMas:31' n = elements(e,:); */
    /*  Nodes of the element 'e' */
    /* 'StiffMas:32' X = nodes(n,:); */
    if (nodes_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_real_T(nodes, &inter_nodes, gpu_nodes);
      nodes_dirtyOnCpu = false;
    }

    if (elements_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_uint32_T(elements, &inter_elements, gpu_elements);
      elements_dirtyOnCpu = false;
    }

    StiffMas_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_nodes, e,
      gpu_elements, *gpu_X);

    /*  Nodal coordinates of the element 'e' */
    /* 'StiffMas:33' ind = repmat(n,8,1); */
    StiffMas_kernel2<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(gpu_elements, e,
      *gpu_ind);

    /*  Index for element 'e' */
    /* 'StiffMas:34' iK(:,:,e) = ind'; */
    /*  Row index storage */
    /* 'StiffMas:35' jK(:,:,e) = ind; */
    /*  Columm index storage */
    /* 'StiffMas:36' Ke(:,:,e) = Hex8scalars(X,c); */
    /*  HEX8SCALARS Compute the element stiffnes matrix for a SCALAR problem in SERIAL computing. */
    /*    HEX8SCALARS(X,c) returns the element stiffness matrix "ke" for an element */
    /*    "e"  in a finite element analysis of scalar problems in a three-dimensional */
    /*    domain computed in a serial manner on the CPU,  where "X" is the nodal */
    /*    coordinates of the element "e" (size 8x3), and "c" the material property */
    /*    (scalar). */
    /*  */
    /*    Examples: */
    /*          X = [-1,-1,-1; 1,-1,-1; 1,1,-1; -1,1,-1; -1,-1,1; 1,-1,1; 1,1,1; -1,1,1] */
    /*          ke = Hex8scalars(X,1) */
    /*   */
    /*    See also HEX8SCALARSAS, HEX8SCALARSAP */
    /*  */
    /*    For more information, see the <a href="matlab: */
    /*    web('https://github.com/fjramireg/StiffMa')">StiffMa</a> web site. */
    /*    Written by Francisco Javier Ramirez-Gil, fjramireg@gmail.com */
    /*    Universidad Nacional de Colombia - Medellin */
    /*  	Modified: 05/12/2019. Version: 1.4. Name changed, Doc improved */
    /*  	Modified: 22/01/2019. Version: 1.3 */
    /*    Created:  30/11/2018. Version: 1.0 */
    /* 'Hex8scalars:24' p = 1/sqrt(3); */
    /*  Gauss points */
    /* 'Hex8scalars:25' r = [-p,p,p,-p,-p,p,p,-p]; */
    /*  Points through r-coordinate */
    /* 'Hex8scalars:26' s = [-p,-p,p,p,-p,-p,p,p]; */
    /*  Points through s-coordinate */
    /* 'Hex8scalars:27' t = [-p,-p,-p,-p,p,p,p,p]; */
    /*  Points through t-coordinate */
    /* 'Hex8scalars:28' ke = zeros(8,8); */
    if (Ke_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_real_T(Ke, &inter_Ke, gpu_Ke);
      Ke_dirtyOnCpu = false;
    }

    if (jK_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_uint32_T(jK, &inter_jK, gpu_jK);
      jK_dirtyOnCpu = false;
    }

    if (iK_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_uint32_T(iK, &inter_iK, gpu_iK);
      iK_dirtyOnCpu = false;
    }

    StiffMas_kernel3<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*gpu_ind, e,
      gpu_Ke, gpu_jK, gpu_iK);
    iK_dirtyOnGpu = true;
    jK_dirtyOnGpu = true;
    Ke_dirtyOnGpu = true;

    /*  Initialize the element stiffness matrix */
    /* 'Hex8scalars:29' for i=1:8 */
    for (b_i = 0; b_i < 8; b_i++) {
      /*  Loop over numerical integration */
      /* 'Hex8scalars:30' ri = r(i); */
      /* 'Hex8scalars:30' si = s(i); */
      /* 'Hex8scalars:30' ti = t(i); */
      /*   Shape function derivatives with respect to r,s,t */
      /* 'Hex8scalars:32' dNdr = (1/8)*[-(1-si)*(1-ti),  (1-si)*(1-ti), (1+si)*(1-ti), -(1+si)*(1-ti),... */
      /* 'Hex8scalars:33'         -(1-si)*(1+ti),  (1-si)*(1+ti), (1+si)*(1+ti), -(1+si)*(1+ti)]; */
      /* 'Hex8scalars:34' dNds = (1/8)*[-(1-ri)*(1-ti), -(1+ri)*(1-ti), (1+ri)*(1-ti),  (1-ri)*(1-ti),... */
      /* 'Hex8scalars:35'         -(1-ri)*(1+ti), -(1+ri)*(1+ti), (1+ri)*(1+ti),  (1-ri)*(1+ti)]; */
      /* 'Hex8scalars:36' dNdt = (1/8)*[-(1-ri)*(1-si), -(1+ri)*(1-si),-(1+ri)*(1+si), -(1-ri)*(1+si),... */
      /* 'Hex8scalars:37'         (1-ri)*(1-si),  (1+ri)*(1-si), (1+ri)*(1+si),  (1-ri)*(1+si)]; */
      /* 'Hex8scalars:38' L = [dNdr; dNds; dNdt]; */
      L[0] = 0.125 * (-(1.0 - dv[b_i]) * (1.0 - dv1[b_i]));
      L[3] = 0.125 * ((1.0 - dv[b_i]) * (1.0 - dv1[b_i]));
      L[6] = 0.125 * ((dv[b_i] + 1.0) * (1.0 - dv1[b_i]));
      L[9] = 0.125 * (-(dv[b_i] + 1.0) * (1.0 - dv1[b_i]));
      L[12] = 0.125 * (-(1.0 - dv[b_i]) * (dv1[b_i] + 1.0));
      L[15] = 0.125 * ((1.0 - dv[b_i]) * (dv1[b_i] + 1.0));
      L[18] = 0.125 * ((dv[b_i] + 1.0) * (dv1[b_i] + 1.0));
      L[21] = 0.125 * (-(dv[b_i] + 1.0) * (dv1[b_i] + 1.0));
      L[1] = 0.125 * (-(1.0 - dv2[b_i]) * (1.0 - dv1[b_i]));
      L[4] = 0.125 * (-(dv2[b_i] + 1.0) * (1.0 - dv1[b_i]));
      L[7] = 0.125 * ((dv2[b_i] + 1.0) * (1.0 - dv1[b_i]));
      L[10] = 0.125 * ((1.0 - dv2[b_i]) * (1.0 - dv1[b_i]));
      L[13] = 0.125 * (-(1.0 - dv2[b_i]) * (dv1[b_i] + 1.0));
      L[16] = 0.125 * (-(dv2[b_i] + 1.0) * (dv1[b_i] + 1.0));
      L[19] = 0.125 * ((dv2[b_i] + 1.0) * (dv1[b_i] + 1.0));
      L[22] = 0.125 * ((1.0 - dv2[b_i]) * (dv1[b_i] + 1.0));
      L[2] = 0.125 * (-(1.0 - dv2[b_i]) * (1.0 - dv[b_i]));
      L[5] = 0.125 * (-(dv2[b_i] + 1.0) * (1.0 - dv[b_i]));
      L[8] = 0.125 * (-(dv2[b_i] + 1.0) * (dv[b_i] + 1.0));
      L[11] = 0.125 * (-(1.0 - dv2[b_i]) * (dv[b_i] + 1.0));
      L[14] = 0.125 * ((1.0 - dv2[b_i]) * (1.0 - dv[b_i]));
      L[17] = 0.125 * ((dv2[b_i] + 1.0) * (1.0 - dv[b_i]));
      L[20] = 0.125 * ((dv2[b_i] + 1.0) * (dv[b_i] + 1.0));
      L[23] = 0.125 * ((1.0 - dv2[b_i]) * (dv[b_i] + 1.0));

      /*  L matrix */
      /* 'Hex8scalars:39' Jac  = L*X; */
      hipMemcpy(gpu_L, &L[0], 192UL, hipMemcpyHostToDevice);
      StiffMas_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_X, *gpu_L, *
        gpu_Jac);

      /*  Jacobian matrix */
      /* 'Hex8scalars:40' detJ = det(Jac); */
      StiffMas_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_Jac, *gpu_x);
      x_dirtyOnGpu = true;
      if (ipiv_dirtyOnCpu) {
        hipMemcpy(gpu_ipiv, &ipiv[0], 3UL, hipMemcpyHostToDevice);
        ipiv_dirtyOnCpu = false;
      }

      StiffMas_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_ipiv);
      ipiv_dirtyOnGpu = true;
      for (j = 0; j < 2; j++) {
        b_c = j << 2;
        jp1j = b_c + 1;
        c_c = 1 - j;
        jA = 0;
        ix = b_c;
        if (x_dirtyOnGpu) {
          hipMemcpy(&x[0], gpu_x, 72UL, hipMemcpyDeviceToHost);
          x_dirtyOnGpu = false;
        }

        smax = fabs(x[b_c]);
        for (jy = 0; jy <= c_c; jy++) {
          ix++;
          s = fabs(x[ix]);
          if (s > smax) {
            jA = jy + 1;
            smax = s;
          }
        }

        if (x[b_c + jA] != 0.0) {
          if (jA != 0) {
            if (ipiv_dirtyOnGpu) {
              hipMemcpy(&ipiv[0], gpu_ipiv, 3UL, hipMemcpyDeviceToHost);
              ipiv_dirtyOnGpu = false;
            }

            ipiv[j] = static_cast<int8_T>(((j + jA) + 1));
            ipiv_dirtyOnCpu = true;
            initAuxVar = j + jA;
            for (jy = 0; jy < 3; jy++) {
              ix = j + jy * 3;
              iy = initAuxVar + jy * 3;
              smax = x[ix];
              x[ix] = x[iy];
              x[iy] = smax;
              x_dirtyOnCpu = true;
            }
          }

          i1 = (b_c - j) + 2;
          for (jA = 0; jA <= i1 - jp1j; jA++) {
            jy = (b_c + jA) + 1;
            x[jy] /= x[b_c];
            x_dirtyOnCpu = true;
          }
        }

        c_c = 1 - j;
        jA = b_c + 5;
        jy = b_c + 3;
        for (iy = 0; iy <= c_c; iy++) {
          smax = x[jy];
          if (x[jy] != 0.0) {
            ix = b_c;
            i1 = jA - 1;
            jp1j = jA - j;
            for (ijA = 0; ijA <= jp1j - i1; ijA++) {
              b_ijA = (jA + ijA) - 1;
              x[b_ijA] += x[ix + 1] * -smax;
              x_dirtyOnCpu = true;
              ix++;
            }
          }

          jy += 3;
          jA += 3;
        }
      }

      detJ = 1.0;
      if (x_dirtyOnCpu) {
        hipMemcpy(gpu_x, &x[0], 72UL, hipMemcpyHostToDevice);
        x_dirtyOnCpu = false;
      }

      hipMemcpy(gpu_detJ, &detJ, 8UL, hipMemcpyHostToDevice);
      StiffMas_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_x, gpu_detJ);
      detJ_dirtyOnGpu = true;
      isodd = false;
      for (jy = 0; jy < 2; jy++) {
        if (ipiv_dirtyOnGpu) {
          hipMemcpy(&ipiv[0], gpu_ipiv, 3UL, hipMemcpyDeviceToHost);
          ipiv_dirtyOnGpu = false;
        }

        if (ipiv[jy] > jy + 1) {
          isodd = !isodd;
        }
      }

      if (isodd) {
        hipMemcpy(&detJ, gpu_detJ, 8UL, hipMemcpyDeviceToHost);
        detJ = -detJ;
        detJ_dirtyOnGpu = false;
      }

      /*  Jacobian's determinant */
      /* 'Hex8scalars:41' B = Jac\L; */
      jA = 1;
      jy = 2;
      jp1j = 3;
      hipMemcpy(&Jac[0], gpu_Jac, 72UL, hipMemcpyDeviceToHost);
      smax = fabs(Jac[0]);
      s = fabs(Jac[1]);
      if (s > smax) {
        smax = s;
        jA = 2;
        jy = 1;
      }

      if (fabs(Jac[2]) > smax) {
        jA = 3;
        jy = 2;
        jp1j = 1;
      }

      Jac[jy - 1] /= Jac[jA - 1];
      Jac[jp1j - 1] /= Jac[jA - 1];
      Jac[jy + 2] -= Jac[jy - 1] * Jac[jA + 2];
      Jac[jp1j + 2] -= Jac[jp1j - 1] * Jac[jA + 2];
      Jac[jy + 5] -= Jac[jy - 1] * Jac[jA + 5];
      Jac[jp1j + 5] -= Jac[jp1j - 1] * Jac[jA + 5];
      if (fabs(Jac[jp1j + 2]) > fabs(Jac[jy + 2])) {
        iy = jy;
        jy = jp1j;
        jp1j = iy;
      }

      hipMemcpy(gpu_Jac, &Jac[0], 72UL, hipMemcpyHostToDevice);
      StiffMas_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(jy, jp1j,
        *gpu_Jac);
      StiffMas_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(jy, jp1j,
        *gpu_Jac);
      StiffMas_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(jp1j, *gpu_Jac,
        jy, *gpu_L, jA, *gpu_B);

      /*  B matrix */
      /* 'Hex8scalars:42' ke = ke + c*detJ*(B'*B); */
      if (detJ_dirtyOnGpu) {
        hipMemcpy(&detJ, gpu_detJ, 8UL, hipMemcpyDeviceToHost);
      }

      smax = c * detJ;
      StiffMas_kernel11<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*gpu_B,
        *b_gpu_B);
      StiffMas_kernel12<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*b_gpu_B, smax,
        gpu_Ke, e, *b_gpu_Ke);
      StiffMas_kernel13<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*b_gpu_Ke, e,
        gpu_Ke);

      /*  Element stiffness matrix */
    }

    /*  Element stiffness matrix compute & storage */
  }

  if (iK_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_uint32_T(iK, &inter_iK);
  }

  if (jK_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_uint32_T(jK, &inter_jK);
  }

  if (Ke_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(Ke, &inter_Ke);
  }

  gpuEmxFree_uint32_T(&inter_elements);
  hipFree(gpu_elements);
  gpuEmxFree_uint32_T(&inter_iK);
  hipFree(gpu_iK);
  gpuEmxFree_uint32_T(&inter_jK);
  hipFree(gpu_jK);
  gpuEmxFree_real_T(&inter_Ke);
  hipFree(gpu_Ke);
  gpuEmxFree_real_T(&inter_nodes);
  hipFree(gpu_nodes);
  hipFree(*gpu_X);
  hipFree(*gpu_ind);
  hipFree(*gpu_L);
  hipFree(*gpu_Jac);
  hipFree(*gpu_x);
  hipFree(*gpu_ipiv);
  hipFree(gpu_detJ);
  hipFree(*gpu_B);
  hipFree(*b_gpu_B);
  hipFree(*b_gpu_Ke);
}

/* End of code generation (StiffMas.cu) */
