
#include <hip/hip_runtime.h>
/*=========================================================================
 *
 ** IndexScalarGPU - Row/column indices of the lower triangular sparse matrix K (SCALAR)
 *
 *
 ** DATA INPUT
 * 			elements[8][nel]      // Conectivity matrix of the mesh
 *
 ** DATA OUTPUT
 *			iK[36*nel]            // Row indices of the lower-triangular part of ke
 *			jK[36*nel]            // Colummn indices of the lower-triangular part of ke
 *
 ** COMPILATION (Terminal)
 * 	 Opt1:	nvcc -ptx IndexScalarsp.cu
 *   Opt2:  nvcc -ptx -v -arch=sm_50 --fmad=false -o IndexScalarsp_cc50.ptx IndexScalarsp.cu
 *
 ** COMPILATION within MATLAB using NVCC
 * 			setenv('MW_NVCC_PATH','/usr/local/cuda-10.1/bin')
 *          setenv('PATH',[getenv('PATH') ':/usr/local/cuda-10.1/bin'])
 *          system('nvcc -ptx IndexScalarsp.cu')
 *
 ** MATLAB KERNEL CREATION (inside MATLAB)
 *			kernel = parallel.gpu.CUDAKernel('IndexScalarsp.ptx', 'IndexScalarsp.cu');
 *
 ** MATLAB KERNEL CONFIGURATION
 *          kernel.ThreadBlockSize = [512, 1, 1];
 *          kernel.GridSize = [ceil(nel/kernel.ThreadBlockSize(1)), 1, 1];
 *
 ** MATLAB CALL
 *			Out = feval(kernel, DATA INPUT + DATA OUTPUT);
 *          [iK, jK] = feval(kernel, elements, nel, gpuArray.zeros(36*nel,1,'uint32'), gpuArray.zeros(36*nel,1,'uint32'));
 *
 ** TRANSFER DATA FROM CPU TO GPU MEMORY (if necessary)
 *			Out_cpu = gather(Out);
 *
 ** This function was developed by:
 *          Francisco Javier Ramirez-Gil
 *          Universidad Nacional de Colombia - Medellin
 *          Department of Mechanical Engineering
 *
 ** Please cite this code as:
 *
 *** Date & version       
 *      Last modified: 07/12/2019. Version 1.4 (added grid stride)
 *      Modified: 21/01/2019, Version 1.3
 *      Created: 30/11/2018. V 1.0
 *
 * ======================================================================*/

template <typename dType>                               // Data type
__global__ void IndexScalarGPU(const dType *elements, const dType nel, dType *iK, dType *jK){
    // CUDA kernel to compute row/column indices of tril(K) (SCALAR)
    
    int tid = blockDim.x * blockIdx.x + threadIdx.x;    // Thread ID
    int gridStride = gridDim.x * blockDim.x;            // Grid stride
    unsigned int e, i, j, temp, idx;                    // General indices
    dType n[8];                                         // DOFs
    
    for (e = tid; e < nel; e += gridStride){            // Parallel computation
        
        // Extract the nodes (DOFs) associated with element 'e'
        for (i=0; i<8; i++) {n[i] = elements[i+8*e];}
        
        temp = 0;
        for (j=0; j<8; j++){
            for (i=j; i<8; i++){
                idx = temp + i + 36*e;
                if (n[i] >= n[j]){
                    iK[idx] = n[i];
                    jK[idx] = n[j];}
                else{
                    iK[idx] = n[j];
                    jK[idx] = n[i];}}
            temp += i-j-1;   }}}

template __global__ void IndexScalarGPU<int>(const int *,
        const int, int *, int *);                               // Indices of data type 'int32'
template __global__ void IndexScalarGPU<unsigned int>(const unsigned int *,
        const unsigned int, unsigned int *, unsigned int *);    // Indices of data type 'uint32'
template __global__ void IndexScalarGPU<long>(const long *,
        const long, long *, long *);                            // Indices of data type 'int64'
template __global__ void IndexScalarGPU<unsigned long>(const unsigned long *,
        const unsigned long, unsigned long *, unsigned long *); // Indices of data type 'uint64'
template __global__ void IndexScalarGPU<double>(const double *,
        const double, double *, double *);                      // Indices of data type 'double'
