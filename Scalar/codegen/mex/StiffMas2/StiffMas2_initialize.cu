/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * StiffMas2_initialize.cu
 *
 * Code generation for function 'StiffMas2_initialize'
 *
 */

/* Include files */
#include "StiffMas2_initialize.h"
#include "StiffMas2.h"
#include "StiffMas2_data.h"
#include "_coder_StiffMas2_mex.h"
#include "rt_nonfinite.h"

/* Function Definitions */
void StiffMas2_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Distrib_Computing_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

/* End of code generation (StiffMas2_initialize.cu) */
