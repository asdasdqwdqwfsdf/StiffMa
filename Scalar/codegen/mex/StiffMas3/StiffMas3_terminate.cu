/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * StiffMas3_terminate.cu
 *
 * Code generation for function 'StiffMas3_terminate'
 *
 */

/* Include files */
#include "StiffMas3_terminate.h"
#include "StiffMas3.h"
#include "StiffMas3_data.h"
#include "_coder_StiffMas3_mex.h"
#include "rt_nonfinite.h"

/* Function Definitions */
void StiffMas3_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void StiffMas3_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

/* End of code generation (StiffMas3_terminate.cu) */
