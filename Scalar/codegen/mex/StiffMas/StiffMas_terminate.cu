/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * StiffMas_terminate.cu
 *
 * Code generation for function 'StiffMas_terminate'
 *
 */

/* Include files */
#include "StiffMas_terminate.h"
#include "StiffMas.h"
#include "StiffMas_data.h"
#include "_coder_StiffMas_mex.h"
#include "rt_nonfinite.h"

/* Function Definitions */
void StiffMas_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void StiffMas_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

/* End of code generation (StiffMas_terminate.cu) */
